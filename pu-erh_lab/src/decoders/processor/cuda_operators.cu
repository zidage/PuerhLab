#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <opencv2/core/mat.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/opencv.hpp>

#include "decoders/processor/cuda_operators.hpp"

namespace puerhlab {
namespace CUDA {
__constant__ float M_const[9];

__global__ void    ApplyColorMatrixKernel(const uchar* srcptr, uchar* dstptr, int rows, int cols,
                                          size_t src_step, size_t dst_step) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows) return;

  const float* src_p = (const float*)(srcptr + y * src_step) + x * 3;
  float*       dst_p = (float*)(dstptr + y * dst_step) + x * 3;

  float        r     = src_p[0];
  float        g     = src_p[1];
  float        b     = src_p[2];

  dst_p[0]           = M_const[0] * r + M_const[1] * g + M_const[2] * b;
  dst_p[1]           = M_const[3] * r + M_const[4] * g + M_const[5] * b;
  dst_p[2]           = M_const[6] * r + M_const[7] * g + M_const[8] * b;
}

void ApplyColorMatrix(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, const cv::Mat& matrix,
                      cv::cuda::Stream& stream) {
  CV_Assert(src.type() == CV_32FC3);
  CV_Assert(matrix.isContinuous() && matrix.rows == 3 && matrix.cols == 3 &&
            matrix.type() == CV_32F);

  if (dst.empty() || dst.size() != src.size() || dst.type() != src.type())
    dst.create(src.size(), src.type());

  hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);

  hipMemcpyToSymbolAsync(HIP_SYMBOL(M_const), matrix.data, 9 * sizeof(float), 0, hipMemcpyHostToDevice,
                          cudaStream);

  dim3 block(16, 16);
  dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

  ApplyColorMatrixKernel<<<grid, block, 0, cudaStream>>>(src.data, dst.data, src.rows, src.cols,
                                                         src.step, dst.step);
}

#define CUDA_CHECK(call)                                                                         \
  do {                                                                                           \
    hipError_t err = call;                                                                      \
    if (err != hipSuccess) {                                                                    \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  } while (0)

// Use __constant__ memory for small arrays that are read by all threads.
// This is faster than global memory.
__constant__ float d_black_level[4];
__constant__ float d_wb_multipliers[4];
__constant__ int   remap[4] = {0, 1, 3, 2};

/**
 * @brief CUDA kernel to perform black level subtraction, white balancing, and white level scaling.
 *
 * @param image The GpuMat data (pointer and step).
 * @param width Image width.
 * @param height Image height.
 * @param white_level_scale The scaling factor (65535.0f / maximum).
 * @param bayer_pattern_offset The offset to determine the starting color of the Bayer pattern.
 * (e.g., for RGGB, this would be the index of R).
 */
__global__ void WhiteBalanceCorrectionKernel(cv::cuda::PtrStep<ushort> image, int width, int height,
                                             float white_level_scale, int bayer_pattern_offset) {
  // Calculate the global x and y coordinates of the pixel for this thread
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;

  // Boundary check to avoid processing out-of-bounds pixels
  if (col >= width || row >= height) {
    return;
  }

  // Determine the color channel (0, 1, 3, 2) for the current pixel based on its position.
  // This standard calculation assumes a 2x2 Bayer pattern (like RGGB, GRBG, etc.).
  // The 'bayer_pattern_offset' helps align to the specific pattern from LibRaw's `idata.filters`.
  // The LibRaw COLOR(row, col) macro can often be simplified to this.
  const int color_idx = remap[(((row % 2) * 2) + (col % 2) + bayer_pattern_offset) % 4];

  // Get a pointer to the current pixel
  ushort*   pixel_ptr = (ushort*)((char*)image.data + row * image.step) + col;

  // --- Start Processing ---

  // 1. Load pixel value and convert to float for processing
  float     pixel_val = static_cast<float>(*pixel_ptr);

  // 2. Black Level Subtraction
  pixel_val -= d_black_level[color_idx];

  pixel_val *= white_level_scale;
  // 3. White Balance Multiplication
  // The multipliers are normalized to the green channel (index 1)
  float       mask   = (color_idx == 0 || color_idx == 2) ? 1.0f : 0.0f;
  const float wb_mul = (d_wb_multipliers[color_idx] / d_wb_multipliers[1]) * mask + (1.0f - mask);
  pixel_val *= wb_mul;

  // 4. White Level Scaling (Normalization)

  // 5. Clamp the result to the valid 16-bit range [0, 65535]
  pixel_val  = fmaxf(0.0f, fminf(65535.0f, pixel_val));

  // 6. Store the final result back to the GpuMat, rounding correctly
  *pixel_ptr = static_cast<ushort>(pixel_val);
}

/**
 * @brief C++ wrapper function to process a raw image on the GPU.
 *
 * @param image The cv::cuda::GpuMat to process (must be CV_16UC1).
 * @param black_level An array of 4 floats for black level correction.
 * @param wb_coeffs An array of 4 floats for white balance.
 * @param maximum The maximum possible pixel value from the sensor.
 * @param apply_wb_and_black_level A flag to control the processing path.
 * @param bayer_offset The starting filter color index (from libraw_data_t.idata.filters).
 */
void WhiteBalanceCorrection(cv::cuda::GpuMat& image, const std::array<float, 4>& black_level,
                            const float* wb_coeffs, float maximum, bool apply_wb_and_black_level,
                            int bayer_offset) {
  // Ensure the input GpuMat has the correct type
  CV_Assert(image.type() == CV_16UC1);
  CV_Assert(maximum > 0);

  const float white_level_scale = 65535.0f / maximum;

  if (apply_wb_and_black_level) {
    // --- Full Processing Path ---

    // 1. Copy black level and WB data from host to GPU's __constant__ memory.
    // This is very fast and efficient for small, read-only data.
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_black_level), black_level.data(), sizeof(float) * 4));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_wb_multipliers), wb_coeffs, sizeof(float) * 4));

    // 2. Define CUDA kernel launch grid dimensions
    const dim3 threads_per_block(16, 16);
    const dim3 num_blocks((image.cols + threads_per_block.x - 1) / threads_per_block.x,
                          (image.rows + threads_per_block.y - 1) / threads_per_block.y);

    // 3. Launch the kernel
    WhiteBalanceCorrectionKernel<<<num_blocks, threads_per_block>>>(
        image, image.cols, image.rows, white_level_scale, bayer_offset);

    // Check for any kernel launch errors (important for debugging)
    CUDA_CHECK(hipGetLastError());
    // Optionally wait for the kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

  } else {
    // --- Simplified Path: Only White Level Scaling ---
    // For this simple operation, OpenCV's built-in function is highly optimized.
    // It performs the operation: image = image * scale.
    // The conversion is done in-place.
    image.convertTo(image, CV_16U, white_level_scale);
  }
}

__global__ void G_FinalGeneration(cv::cuda::PtrStep<float> raw, cv::cuda::PtrStep<float> G_final,
                                  cv::cuda::PtrStep<float> R_final,
                                  cv::cuda::PtrStep<float> B_final, int width, int height) {
  // Kernel implementation goes here
  const int x = blockIdx.x * blockDim.x + threadIdx.x; // col
  const int y = blockIdx.y * blockDim.y + threadIdx.y; // row

  // Boundary check to avoid processing out-of-bounds pixels
  if (y >= height - 2 || x >= width - 2 || y < 2 || x < 2) {
    return;
  }

  const int color_idx = remap[(((y % 2) * 2) + (x % 2)) % 4];
  if (color_idx == 0 || color_idx == 2) {
    float center  = raw(y, x);
    float h_avg   = 0.5f * (raw(y, x - 1) + raw(y, x + 1));
    float h_diff  = 0.25f * (2.0f * center - raw(y, x - 2) - raw(y, x + 2));

    float v_avg   = 0.5f * (raw(y - 1, x) + raw(y + 1, x));
    float v_diff  = 0.25f * (2.0f * center - raw(y - 2, x) - raw(y + 2, x));

    float Dh      = std::abs(raw(y, x - 1) - raw(y, x + 1));
    float Dv      = std::abs(raw(y - 1, x) - raw(y + 1, x));

    G_final(y, x) = (Dh < Dv) ? (h_avg + h_diff) : (v_avg + v_diff);

    R_final(y, x) = color_idx == 0 ? center : 0.0f;
    B_final(y, x) = color_idx == 2 ? center : 0.0f;
  }
}

__global__ void R_B_FinalGeneration(cv::cuda::PtrStep<float> raw, cv::cuda::PtrStep<float> G_final,
                                    cv::cuda::PtrStep<float> R_final,
                                    cv::cuda::PtrStep<float> B_final, int width, int height) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x; // col
  const int y = blockIdx.y * blockDim.y + threadIdx.y; // row

  // Boundary check to avoid processing out-of-bounds pixels
  if (y >= height - 1 || x >= width - 1 || y < 1 || x < 1) {
    return;
  }

  const int color_idx = remap[(((y % 2) * 2) + (x % 2)) % 4];
  if (color_idx != 0) {
    float estimate_R = 0.0f;

    if (color_idx == 1) {
      float left  = R_final(y, x - 1) - G_final(y, x - 1);
      float right = R_final(y, x + 1) - G_final(y, x - 1);
      estimate_R  = G_final(y, x) + 0.5f * (left + right);
    } else if (color_idx == 3) {
      float up   = R_final(y - 1, x) - G_final(y - 1, x);
      float down = R_final(y + 1, x) - G_final(y + 1, x);
      estimate_R = G_final(y, x) + 0.5f * (up + down);
    } else if (color_idx == 2) {
      // At a B pixel: R is on diagonals
      float d1   = R_final(y - 1, x - 1) - G_final(y - 1, x - 1);
      float d2   = R_final(y - 1, x + 1) - G_final(y - 1, x + 1);
      float d3   = R_final(y + 1, x - 1) - G_final(y + 1, x - 1);
      float d4   = R_final(y + 1, x + 1) - G_final(y + 1, x + 1);
      estimate_R = G_final(y, x) + 0.25f * (d1 + d2 + d3 + d4);
    } else {
      float left  = R_final(y, x - 1) - G_final(y, x - 1);
      float right = R_final(y, x + 1) - G_final(y, x + 1);
      float up    = R_final(y - 1, x) - G_final(y - 1, x);
      float down  = R_final(y + 1, x) - G_final(y + 1, x);
      estimate_R  = G_final(y, x) + 0.25f * (left + right + up + down);
    }
    R_final(y, x) = estimate_R;
  }

  if (color_idx != 2) {
    float estimate_B = 0.0f;
    if (color_idx == 3) {
      float left  = B_final(y, x - 1) - G_final(y, x - 1);
      float right = B_final(y, x + 1) - G_final(y, x + 1);
      estimate_B  = G_final(y, x) + 0.5f * (left + right);
    } else if (color_idx == 1) {
      float up   = B_final(y - 1, x) - G_final(y - 1, x);
      float down = B_final(y + 1, x) - G_final(y + 1, x);
      estimate_B = G_final(y, x) + 0.5f * (up + down);
    } else if (color_idx == 0) {
      // At an R pixel: B is on diagonals
      float d1   = B_final(y - 1, x - 1) - G_final(y - 1, x - 1);
      float d2   = B_final(y - 1, x + 1) - G_final(y - 1, x + 1);
      float d3   = B_final(y + 1, x - 1) - G_final(y + 1, x - 1);
      float d4   = B_final(y + 1, x + 1) - G_final(y + 1, x + 1);
      estimate_B = G_final(y, x) + 0.25f * (d1 + d2 + d3 + d4);
    } else {
      float left  = B_final(y, x - 1) - G_final(y, x - 1);
      float right = B_final(y, x + 1) - G_final(y, x + 1);
      float up    = B_final(y - 1, x) - G_final(y - 1, x);
      float down  = B_final(y + 1, x) - G_final(y + 1, x);
      estimate_B  = G_final(y, x) + 0.25f * (left + right + up + down);
    }
    B_final(y, x) = estimate_B;
  }
}

void BayerRGGB2RGB_AHD(cv::cuda::GpuMat& image) {
  const dim3 threads_per_block(16, 16);
  const dim3 num_blocks((image.cols + threads_per_block.x - 1) / threads_per_block.x,
                        (image.rows + threads_per_block.y - 1) / threads_per_block.y);

  image.convertTo(image, CV_32FC1, 1.0f / 65535.0f);

  cv::cuda::GpuMat G_final = image.clone();
  cv::cuda::GpuMat R_final;
  R_final.create(image.size(), CV_32FC1);
  cv::cuda::GpuMat B_final;
  B_final.create(image.size(), CV_32FC1);

  G_FinalGeneration<<<num_blocks, threads_per_block>>>(image, G_final, R_final, B_final, image.cols,
                                                       image.rows);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  R_B_FinalGeneration<<<num_blocks, threads_per_block>>>(image, G_final, R_final, B_final,
                                                         image.cols, image.rows);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  std::vector<cv::cuda::GpuMat> channels = {R_final, G_final, B_final};
  cv::cuda::merge(channels, image);
}
};  // namespace CUDA
};  // namespace puerhlab
